#include "hip/hip_runtime.h"
#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"

texture<float4, 3, hipReadModeElementType> noiseTex;

__constant__ SimParams params;

__device__
float3 noise3D(float3 p)
{
	float4 n=tex3D(noiseTex,p.x,p.y,p.z);
	return make_float3(n.x,n.y,n.z);
}

__device__
float3 fractalSum3D(float3 p,int octaves,float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += noise3D(p*freq)*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;	
}

__device__
float3 turbulence3D(float3 p, int octaves, float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += fabs(noise3D(p*freq))*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

__global__ void
integrateD(float4* newPos,float4* newVel,
		   float4* oldPos,float4* oldVel,
		   float deltaTime,
		   int numParticles)
{
	int index=__mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index>=numParticles) return;
	
	volatile float4 posData=oldPos[index];
	volatile float4 velData=oldVel[index];
	
	float3 pos=make_float3(posData.x,posData.y,posData.z);
	float3 vel=make_float3(velData.x,velData.y,posData.z);
	
	float age=posData.w;
	float lifetime=velData.w;
	if(age<lifetime)
	{
		age+=deltaTime;
	}
	else
	{
		age=lifetime;
	}
	
	vel+=params.gravity*deltaTime;
	
	float3 noise=noise3D(pos*params.noiseFreq+params.time*params.noiseSpeed);
	vel+=noise*params.noiseAmp;
	
	pos+=vel*deltaTime;
	
	vel*=params.globalDamping;
	
	newPos[index]=make_float4(pos,age);
	newVel[index]=make_float4(vel,velData.w);
}

__global__ void calcDepthD(float4* pos,float* keys,uint *indices,float3 vector,int numParticles)
{
	uint index=__mul24(blockIdx.x,blockDim.x)+threadIdx.x;
	if(index>=numParticles) return ;
	
	volatile float4 p = pos[index];
	float key = -dot(make_float3(p.x,p.y,p.z),vector);
	
	keys[index]=key;
	indices[index]=index;
}

#endif