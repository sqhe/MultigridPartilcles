#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* 
 * CUDA Device code for particle simulation.
 */

#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"

texture<float4, 3, hipReadModeElementType> noiseTex;

// simulation parameters
__constant__ SimParams params;

// look up in 3D noise texture
__device__
float3 noise3D(float3 p)
{
    float4 n = tex3D(noiseTex, p.x, p.y, p.z);
    return make_float3(n.x, n.y, n.z);
}

__device__
float3 fractalSum3D(float3 p, int octaves, float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += noise3D(p*freq)*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

__device__
float3 turbulence3D(float3 p, int octaves, float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += fabs(noise3D(p*freq))*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

// integrate particle attributes
__global__ void
integrateD(float4* newPos, float4* newVel, 
           float4* oldPos, float4* oldVel, 
           float deltaTime,
		   int numParticles)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index >= numParticles) return;

	volatile float4 posData = oldPos[index];	// ensure coalesced reads
    volatile float4 velData = oldVel[index];

    float3 pos = make_float3(posData.x, posData.y, posData.z);
    float3 vel = make_float3(velData.x, velData.y, velData.z);
    
    // update particle age
	float age = posData.w;	
	float lifetime = velData.w;
	if (age < lifetime) {
		age += deltaTime;
	} else {
	    age = lifetime;
    }

    // apply accelerations
    //vel += params.gravity * deltaTime;

    // apply procedural noise
    //float3 noise = noise3D(pos*params.noiseFreq + params.time*params.noiseSpeed);
    //vel += noise * params.noiseAmp;

    // new position = old position + velocity * deltaTime
    pos += vel * deltaTime;

    //vel *= params.globalDamping;
    
    //if ((index%10)==1)
    //vel *=2;
    //else if((index%10)==2)
    //vel *=3;
    //else if((index%10)==3)
    //vel *=4;
    //else if((index%10)==4)
    //vel *=5;
    //else if((index%10)==5)
    //vel *=6;
    //else if((index%10)==6)
    //vel *=7;
    //else if((index%10)==7)
    //vel *=8;
    //else if((index%10)==8)
    //vel *=9;
    //else if((index%10)==9)
    //vel *=10;    

    // store new position and velocity
    newPos[index] = make_float4(pos, age);
    newVel[index] = make_float4(vel, velData.w);
}

// calculate sort depth for each particle
__global__ void calcDepthD(float4* pos, float* keys, uint *indices, float3 vector, int numParticles)
{
	uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index >= numParticles) return;

	volatile float4 p = pos[index];
	float key = -dot(make_float3(p.x, p.y, p.z), vector);        // project onto sort vector
	
	keys[index] = key;
	indices[index] = index;
}

#endif
