#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include "particles_kernel.cu"
#include "ParticleSystem.cuh"

extern "C"
{

hipArray *noiseArray;

void initCuda(bool bUseGL)
{
	if (bUseGL) {
	    cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
	} else {
		hipSetDevice( cutGetMaxGflopsDeviceId() );
	}
}

void setParameters(SimParams *hostParams)
{
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

int iDivUp(int a, int b)
{
	return (a%b != 0)? (a/b +1) : (a/b) ;
}

void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
	numThreads=min(blockSize, n);
	numBlocks=iDivUp(n,numThreads);
}

inline float frand()
{
	return rand()/(float) RAND_MAX;
}

void createNoiseTexture(int w,int h,int d)
{
    hipExtent size = make_hipExtent(w, h, d);
    uint elements = (uint) size.width*size.height*size.depth;

    float *volumeData = (float *)malloc(elements*4*sizeof(float));
    float *ptr = volumeData;
    for(uint i=0; i<elements; i++) {
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
    }


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    cutilSafeCall( hipMalloc3DArray(&noiseArray, &channelDesc, size) );

    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr   = make_hipPitchedPtr((void*)volumeData, size.width*sizeof(float4), size.width, size.height);
    copyParams.dstArray = noiseArray;
    copyParams.extent   = size;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );

    free(volumeData);

    // set texture parameters
    noiseTex.normalized = true;                      // access with normalized texture coordinates
    noiseTex.filterMode = hipFilterModeLinear;      // linear interpolation
    noiseTex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    noiseTex.addressMode[1] = hipAddressModeWrap;
    noiseTex.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(noiseTex, noiseArray, channelDesc));
}

void
integrateSystem(float4 *oldPos, float4 *newPos,
				float4 *oldVel, float4 *newVel,
				float deltaTime,
				int numParticles)
{
	int numThreads,numBlocks;
	computeGridSize(numParticles, 256, numBlocks,numThreads);
	
	integrateD<<<numBlocks,numThreads>>>(newPos,newVel,
					 oldPos,oldVel,
					 deltaTime,
					 numParticles);
					 
	cutilCheckMsg("Kernel execution failed");
}
				
void
calcDepth(float4* pos,
		  float* keys,
		  uint* indices,
		  float3 sortVector,
		  int numParticles)
{
	int numThreads,numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	
	calcDepthD<<<numBlocks,numThreads>>>(pos,keys,indices,sortVector,numParticles);
	cutilCheckMsg("calcDepthD execution failed");
}

}